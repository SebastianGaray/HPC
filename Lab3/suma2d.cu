
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <unistd.h>
#include <omp.h>
#include <sys/times.h>
#include <time.h>
#include <sys/times.h>
#include <stdio.h>

// Funcion que calcula la suma de vecindades en cpu
void suma2d_cpu(float *A, float *B, int N, int V){
  for(int posX = 0; posX < N; posX++){
    for(int posY = 0; posY < N; posY++){
      for(int i = (posX-V); i<=(posX+V); i++){
        for(int j = (posY-V); j<=(posY+V); j++){
          if(!(i < 0 || i >= N || j < 0 || j >= N)){
            B[posX*N+posY] = B[posX*N+posY] + A[i*N+j];
          }
        }
      } 
    } 
  }
}

// Memoria global
__global__ void suma2d(float *A, float *B, int N, int V)
{
  // Se obtiene la pos x de la hebra
	int posX = blockDim.x*blockIdx.x + threadIdx.x;
	// Se obtiene la pos y de la hebra
	int posY = blockDim.y*blockIdx.y + threadIdx.y; 
	for(int i = (posX-V); i<=(posX+V); i++){
		for(int j = (posY-V); j<=(posY+V); j++){
			if(!(i < 0 || i >= N || j < 0 || j >= N)){
				B[posX*N+posY] = B[posX*N+posY] + A[i*N+j];
			}
		}
	}

}	

int main(int argc, char **argv){
	int N, V, Bs, aux;
    while ((aux = getopt (argc, argv, ":N:B:V:")) != -1){
		switch (aux){
			case 'N':
				N = atoi(optarg);
				break;
            case 'V':
				V = atoi(optarg);
				break;
			case 'B':
				Bs = atoi(optarg);
				break;
			case '?':
                fprintf(stderr, "ERROR, Opcion invalida: -%c\n", optopt);
                return EXIT_FAILURE;
			case ':':
                fprintf(stderr, "ERROR, Falta el argumento de la opcion: -%c\n", optopt);
                return EXIT_FAILURE;
		}
	}
    int size = N*N*sizeof(int);
	srand(time(NULL)); 
    //MEMORIA HOST
	float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
	// SE INICIALIZA
    for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            A[i*N+j] = (float)rand()/RAND_MAX;
            B[i*N+j] = 0;
        }
    }

	//COPIA DESDE MEMORIA A DEVICE
    float *d_a;
    float *d_b;
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
    hipError_t err = hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
    if(err!=hipSuccess) {
        printf("Error al copiar hacia device arreglo A: %s\n", hipGetErrorString(err));
    }
    err = hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);
    if(err!=hipSuccess) {
        printf("Error al copiar hacia device arreglo B: %s\n", hipGetErrorString(err));
    }
  
    //TAMAÑO BLOQUES
    dim3 gridSize = dim3(N/Bs, N/Bs);
    dim3 blockSize = dim3(Bs, Bs);

	//LLAMADO A KERNEL
    // Nota: Se comenta la medicion del tiempo del programa
    //cudaEvent_t start, stop;
    //float elapsedTime = 0;
    //cudaEventCreate(&start);
    //cudaEventCreate(&stop);
    //cudaEventRecord(start, 0);
	suma2d<<<gridSize, blockSize>>>(d_a, d_b, N, V);
    //cudaEventRecord(stop, 0);
    //cudaEventSynchronize(stop);
    //cudaEventElapsedTime(&elapsedTime, start, stop);
    //printf("El tiempo de ejecucion en GPU fue de: %f milisegundos\n", elapsedTime);
  
    //COPIA RESULTADO DESDE DEVICE A HOST
    err = hipMemcpy(B, d_b, size, hipMemcpyDeviceToHost);
    if(err!=hipSuccess) {
        printf("Error al copiar hacia host arreglo B: %s\n", hipGetErrorString(err));
    }
    printf("Resultado CUDA:\n");
        for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            printf("%f ", B[i*N+j]);
        }
        printf("\n");
    }
	hipFree(d_a);
    hipFree(d_b);
  
    ///////
    //CPU//
    ///////
    // Inicializacion del vector para GPU
    float *B_GPU = (float *)malloc(size);
    for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            B_GPU[i*N+j] = 0;
        }
    }
    // Nota: Se comenta la medicion del tiempo del programa
    //clock_t t; 
    //t = clock(); 
    suma2d_cpu(A, B_GPU, N,V);
    //t = clock() - t; 
    //double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 
	//printf("El tiempo de ejecucion en CPU fue de: %f milisegundos\n", time_taken*1000);
    printf("Resultado CPU:\n");
    for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            printf("%f ", B_GPU[i*N+j]);
        }
        printf("\n");
    }
    free(A);
    free(B);
    free(B_GPU);
    return 0;

}