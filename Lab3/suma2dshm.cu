
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <ctype.h>
#include <unistd.h>
#include <omp.h>
#include <sys/times.h>
#include <time.h>
#include <sys/times.h>
#include <stdio.h>
const int threadsPerBlock = 16;

// Funcion que calcula la suma de vecindades en cpu
void suma2d_cpu(float *A, float *B, int N, int V){
  for(int posX = 0; posX < N; posX++){
    for(int posY = 0; posY < N; posY++){
      for(int i = (posX-V); i<=(posX+V); i++){
        for(int j = (posY-V); j<=(posY+V); j++){
          if(!(i < 0 || i >= N || j < 0 || j >= N)){
            B[posX*N+posY] = B[posX*N+posY] + A[i*N+j];
          }
        }
      } 
    } 
  }
}



// Memoria compartida
__global__ void suma2dshm(float *A, float *B, int N, int V)
{
	// Se define la variable compartida temp, con el tamaño del bloque
  __shared__ float temp[threadsPerBlock*threadsPerBlock];
  // Se obtiene la pos x de la hebra
	int posX = blockDim.x*blockIdx.x + threadIdx.x;
 
	// Se obtiene la pos y de la hebra
	int posY = blockDim.y*blockIdx.y + threadIdx.y; 

	for(int i = (posX-V); i<=(posX+V); i++){
		for(int j = (posY-V); j<=(posY+V); j++){
			if(!(i < 0 || i >= N || j < 0 || j >= N)){
        temp[threadIdx.x*threadsPerBlock+threadIdx.y] = temp[threadIdx.x*threadsPerBlock+threadIdx.y] + A[i*N+j];
			}
		}
	}
	
	B[posX*N+posY] = temp[threadIdx.x*threadsPerBlock+threadIdx.y];	
}

int main(int argc, char **argv){
	int N, V, Bs, aux;;
    while ((aux = getopt (argc, argv, ":N:B:V:")) != -1){
		switch (aux){
			case 'N':
				N = atoi(optarg);
				break;
            case 'V':
				V = atoi(optarg);
				break;
			case 'B':
				Bs = atoi(optarg);
				break;
			case '?':
                fprintf(stderr, "ERROR, Opcion invalida: -%c\n", optopt);
                return EXIT_FAILURE;
			case ':':
                fprintf(stderr, "ERROR, Falta el argumento de la opcion: -%c\n", optopt);
                return EXIT_FAILURE;
		}
    }
    int size = N*N*sizeof(float);
	srand(time(NULL)); 
    //MEMORIA HOST
	float *A = (float *)malloc(size);
    float *B = (float *)malloc(size);
    // SE INICIALIZA
    for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            A[i*N+j] = (float)rand()/RAND_MAX;
            B[i*N+j] = 0;
        }
    }
	//COPIA DESDE MEMORIA A DEVICE
    float *d_a;
    float *d_b;
	hipMalloc((void **) &d_a, size);
	hipMalloc((void **) &d_b, size);
    hipError_t err = hipMemcpy(d_a, A, size, hipMemcpyHostToDevice);
    if(err!=hipSuccess) {
        printf("Error al copiar hacia device arreglo A: %s\n", hipGetErrorString(err));
    }
    err = hipMemcpy(d_b, B, size, hipMemcpyHostToDevice);
    if(err!=hipSuccess) {
        printf("Error al copiar hacia device arreglo B: %s\n", hipGetErrorString(err));
    }
    
    //TAMAÑO BLOQUES
    dim3 gridSize = dim3(N/Bs, N/Bs);
    dim3 blockSize = dim3(Bs, Bs);
    
	//LLAMADO A KERNEL
    // Nota: Se comenta la medicion del tiempo del programa
    //cudaEvent_t start, stop;
    //float elapsedTime = 0;
    //cudaEventCreate(&start);
    //cudaEventCreate(&stop);
    //cudaEventRecord(start, 0);
	suma2dshm<<<gridSize, blockSize>>>(d_a, d_b, N, V);
    //cudaEventRecord(stop, 0);
    //cudaEventSynchronize(stop);
    //cudaEventElapsedTime(&elapsedTime, start, stop);
	//printf("El tiempo de ejecucion en GPU fue de: %f segundos\n", elapsedTime/1000);
    //COPIA RESULTADO DESDE DEVICE A HOST
    err = hipMemcpy(B, d_b, size, hipMemcpyDeviceToHost);
    if(err!=hipSuccess) {
        printf("Error al copiar hacia host arreglo B: %s\n", hipGetErrorString(err));
    }
    printf("Resultado CUDA:\n");
    for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            printf("%f ", B[i*N+j]);
        }
        printf("\n");
    }
	hipFree(d_a);
    hipFree(d_b);
     
    ///////
    //CPU//
    ///////

    float *B_GPU = (float *)malloc(size);
    for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            B_GPU[i*N+j] = 0;
        }
    }
    // Nota: Se comenta la medicion del tiempo del programa
    // clock_t t; 
    // t = clock(); 
    suma2d_cpu(A, B_GPU, N,V);
    //t = clock() - t; 
    //double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 
	//printf("El tiempo de ejecucion en CPU fue de: %f segundos\n", time_taken);
    printf("Resultado CPU:\n");
    for(int i = 0; i<N; i++){
        for(int j = 0; j<N; j++){
            printf("%f ", B_GPU[i*N+j]);
        }
        printf("\n");
    }
    free(A);
    free(B);
    free(B_GPU);
    return 0;

}
