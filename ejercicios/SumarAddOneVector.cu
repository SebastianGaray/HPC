#include "hip/hip_runtime.h"
global_ void sumreduction(int *A, int N, int *sum ) {

// Declare memoria compartida para el bloque
_shared_ float temp[T];

int tid = blockDim.x*blockIDx.x + threadIdx.x; // ID global de la hebra

// Cargar bloque de memoria compartida
temp[tid] = A[tid]; 

// Sincronizar a que todas hayan terminado
__syncthreads(); // TB barrier

// Reduccion iterativa dentro del bloque
while(N > 1){
    if( tid < N/2 ) 
        temp[tid] = temp[tid] + temp[tid + N/2];
    __syncthreads();
    if(tid == 0)
        N = N/2;
}

// Reduccion total a memoria global sum
if(tid == 0)
    atomicAdd(sum, temp[0]); //atomic add to global memory
}

_host_ main(){
    
}