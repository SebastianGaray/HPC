#include "hip/hip_runtime.h"
 

//a) Numero de hebras = N

__global__ simpleHist(int *A, int N, int *H, int M) {

	int gid = blockId.x*blockDim.x + threadId.x; // global ID

	atomicAdd(&H[A[gid]], 1);  // muy importante. Es Sección Crítica
}

// b) Numero de hebras > N

__global__ simpleHist(int *A, int N, int *H, int M) {

	int gid = blockId.x*blockDim.x + threadId.x; // global ID

	if (gid < N)  // por definición 0 <= H[j] <= M-1
	   atomicAdd(&H[A[gid]], 1);  // muy importante. Es Sección Crítica
}

// c) Numero de hebras < N

__global__ simpleHist(int *A, int N, int *H, int M) {

	int gid = blockId.x*blockDim.x + threadId.x; // global ID
	int fulloffset = gridDim.x*blockDim.x;       // numero total de hebras
	
	int i = gid;
	while (i < N)  {
	   atomicAdd(&H[A[gid]], 1);  // muy importante. Es Sección Crítica
	   i += fulloffset;           // full stride asegura única posición en A[]
	}
}


//2) Con shared mem

//*** Solucion 1 ***
// --------------------

_global__ simpleHist(int *A, int N, int *H, int M) {

	int gid = blockId.x*blockDim.x + threadId.x; // global ID
	int lid = threadId.x // local ID

	__shared__ int sharedH[M] // se acepta mem dinámica

	sharedH[lid] = 0; // muy importante. H[] está inicializado pero no sharedH[]	
	atomicAdd(&sharedH[A[gid]], 1); // Sección crítica

	__synchthreads();

	if (lid == 0) // solo hebra 0 del bloque reduce
	   for (int i=0; i < M; i++)
		atomicAdd(&H[i], sharedH[i]);
}



/*
*** Solucion 2 ***
--------------------
*/
__global__ simpleHist(int *A, int N, int *H, int M) {

	int gid = blockId.x*blockDim.x + threadId.x; // global ID
	int lid = threadId.x // local ID

	__shared__ int sharedA[blockdim.x] // se acepta mem dinámica
	
	sharedA[lid] = A[gid]; // Cargar memoria compartida!!!!

	int Hlocal[M]; // El histograma es local

	if (lid == 0) 
	   for (int i=0; i < M; i++) Hlocal[i] = 0; // muy importante. H[] está inicializado pero no Hlocal[];

	atomicAdd(&Hlocal[sharedA[lid]], 1);  // muy importante. Es Sección Crítica

	__synchthreads();

	if (lid == 0) // solo hebra 0 del bloque reduce
	   for (int i=0; i < M; i++)
		atomicAdd(&H[i], Hlocal[i]);
}


/*
3) OpenMP


*** Solución 1 ***
------------------
*/

void Hist(int *A, int N, int *H, int M, int n) {


        #pragma omp parallel num_threads(n)   // por defecto todo compartido
	{
	    #pragma omp for 
	    for (int i=0; i < N; i++)
		#pragma omp atomic
			H[A[i]]++;   // Esto es mas ficiente que sección crítica
	}	
}

/*

*** Solución 2 ***
------------------

*/
void Hist(int *A, int N, int *H, int M, int n) {


        #pragma omp parallel num_threads(n)   // por defecto todo compartido
	{
	    #pragma omp for 
	    for (int i=0; i < N; i++)
		#pragma omp critical
			H[A[i]]++;   
	}	
}

/*
4) Amdahl's Law

a)
Esta ley dice que el speedup de una aplicación paralela, DE CARGA FIJA, está limitado
por la porción secuencial de dicha aplicación. Es decir no importa si tenemos infinito número de 
procesadores  para acelerar la aplicación, el speedup siempre está cotado, de la siguiente
manera

Sea f la porción de trabajo secuencial de una aplicación, es decir aquella parte del código
que no puede paralelizarse, y se 1-f la parte paralelizable. Luego, según Amdahl el Speedup
está acotado por

S <= 1/f 

independientemente del número de procesadores.

b) Según lo anterior f = 1/4 y 1-f = 3/4. Si se usa un procesador 3 veces más rápico, entonces
la porción paralelizable se reduce a 

		1 x 3       1
                -   -   =   -
		3   4       4

Luego, la porciones de la aplicación son  1/4 + 1/  = 1/2.  Por lo tanto, el Speedup es 2, es decir
se ejecuta dos veces más rápido.

c) Según a)

	    1
        S=  -   = 4
           1/4


*/ 